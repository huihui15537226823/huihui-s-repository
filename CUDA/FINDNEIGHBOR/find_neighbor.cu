#include "hip/hip_runtime.h"
#include <stdio.h>
#include <
//构建邻居列表的函数
//real为int 或者double
//这个背景是测定原子的邻居,其中N是总的原子数
//MN是每个原子的最多邻居数
//判断两个原子是否为邻居的截断距离平方cutoff_square
//整型数组NN,NN[n]是第n个粒子的邻居个数
//整型数组NL,数组长度为N*MN,NL[n*MN+k]是第n个粒子的第k个邻居的坐标
//x y分别记录每个原子的坐标
void find_neighbor(int *NN,int *NL,const real *x,const real *y){
    for(int n=0;n<N;n++){
        NN[n]=0;
    }
    for(int n1=0;n1<N;++n1){
        real x1=x[n1];
        real y1=y[n1];
        for(int n2=n1+1;n2<N;++n2){
            real x12=x[n2]-x1;
            real y12=y[n2]-y1;
            real distance_square=x12*x12+y12*y12;
            if(distance_square<cutoff_square){
                NL[n1*MN+NN[n1]++]=n2;
                NL[n2*MN+NN[n2]++]=n1;
            }
        }
    }
}

//利用原子操作的cuda版本
void __global__ find_neighbor_atomic (int *d_NL,const real *d_x,const real *d_y,
const int N,const real cutoff_square){
    int n1=blockIdx.x*blockDim.x+threadIdx.x;
    if(n1<N){
        d_NN[n1]=0;
        real x1=d_x[n1];
        real y1=d_y[n1];
        for(int n2=n1+1;n2<N;++n2){
            real x12=d_x[n2]-x1;
            real y12=d_y[n2]-y1;
            real distance_square=x12*x12+y12*y12;
            if(distance_square<cutoff_square){
                d_NL[n1*MN+atomicAdd(&d_NN[n1],1)]=n2;
                d_NL[n2*MN+atomicAdd(&d_NN[n2],1)]=n1;
            }
        }
    }
}