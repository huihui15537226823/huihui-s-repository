
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
    int deviceId;
    hipGetDevice(&deviceId);

    // �����豸���Խṹ�岢��ȡ�豸����
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);

    // �����豸��Ϣ
    int computeCapabilityMajor = props.major;
    int computeCapabilityMinor = props.minor;
    int multiProcessorCount = props.multiProcessorCount;
    int warpSize = props.warpSize;
    int maxThreadsPerBlock = props.maxThreadsPerBlock;
    int maxThreadsPerMultiProcessor = props.maxThreadsPerMultiProcessor;

    // ����Warp�����Ϣ
    int warpsPerSM = maxThreadsPerMultiProcessor / warpSize;
    int totalWarps = warpsPerSM * multiProcessorCount;

    // ��ȡά������
    int maxThreadsDim[3] = {props.maxThreadsDim[0],
                                props.maxThreadsDim[1],
                                props.maxThreadsDim[2]};

    int maxGridSize[3] = {props.maxGridSize[0],
                               props.maxGridSize[1],
                               props.maxGridSize[2]};

    // ��ȡ�ڴ���Ϣ
    size_t totalGlobalMem = props.totalGlobalMem;
    size_t sharedMemPerBlock = props.sharedMemPerBlock;

    // ��ӡ�����豸��Ϣ
    printf("\n===== CUDA �豸���� =====\n");
    printf("\n������Ϣ:\n");
    printf("�豸����: %s\n", props.name);
    printf("�豸ID: %d\n", deviceId);
    printf("��������: %d.%d\n", computeCapabilityMajor, computeCapabilityMinor);

    printf("\n��������Ϣ:\n");
    printf("SM����: %d\n", multiProcessorCount);
    printf("ÿ��SM������߳���: %d\n", maxThreadsPerMultiProcessor);
    printf("ÿ���������߳���: %d\n", maxThreadsPerBlock);

    printf("\nWarp��Ϣ:\n");
    printf("Warp��С: %d\n", warpSize);
    printf("ÿ��SM��Warp����: %d\n", warpsPerSM);
    printf("��Warp����: %d\n", totalWarps);

    printf("\nά������:\n");
    printf("����߳�ά��: (%d, %d, %d)\n",
           maxThreadsDim[0], maxThreadsDim[1], maxThreadsDim[2]);
    printf("�������ά��: (%d, %d, %d)\n",
           maxGridSize[0], maxGridSize[1], maxGridSize[2]);

    printf("\n�ڴ���Ϣ:\n");
    printf("ȫ���ڴ�: %.2f GB\n", (float)totalGlobalMem / (1024*1024*1024));
    printf("ÿ�鹲���ڴ�: %zu KB\n", sharedMemPerBlock / 1024);

    printf("\n��������:\n");
    printf("��󲢷��߳���: %d\n", multiProcessorCount * maxThreadsPerMultiProcessor);
    printf("��󲢷�Warp��: %d\n", totalWarps);

    return 0;
}