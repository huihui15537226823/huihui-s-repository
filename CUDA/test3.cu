
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Initialize array values on the host.
 */
// �������˳�ʼ�����飺������Ԫ������Ϊ������ֵ
void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;  // ÿ��Ԫ�ص�ֵ����������
  }
}

__global__ void doubleElements(int *a, int N)
{
  int i;
  // ����ȫ���߳�����
  i = blockIdx.x * blockDim.x + threadIdx.x;
  // ȷ����Խ��
  if (i < N)
  {
    a[i] *= 2;  // ��Ԫ��ֵ����
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    // ���ÿ��Ԫ���Ƿ����������ֵ������
    if (a[i] != i*2) return false;
  }
  return true;
}

int main(){
    int N=100;
    int *a;
    size_t size=N * sizeof(int);
    //����ͳһ�ڴ�,ʹcpu gpu ���ܷ���
    hipMallocManaged(&a,size);

    init(a,N);

    // ����CUDA�˺�����ִ������
    size_t threads_per_block = 10;  // ÿ����10���߳�
    size_t number_of_blocks = 10;  // �ܹ�10����

    // �����˺��������д�������
    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    // �ȴ�GPU�������
    hipDeviceSynchronize();

    // ��֤���
    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

    // �ͷ�ͳһ�ڴ�
    hipFree(a);
}