
#include <hip/hip_runtime.h>
#include <stdio.h>
//CUDA处理宏,包装cuda函数调用
#define gpuErrchk(ans) {gpuAssert((ans),__FILE__, __LINE__);}

//错误处理辅助函数
inline void gpuAssert(hipError_t code,const char *file,int line){
    if(code!=hipSuccess){
        fprintf(stderr,"GPU错误:%s\n 文件:%s\n 行号:%d\n", hipGetErrorString(code),file,line);
        exit(code);
    }
}

// CPU函数：用指定值初始化数组
void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

// GPU核函数：并行执行向量加法
// 使用网格跨度循环处理大数组
__global__ void addVectorsInto(float *result,float *a,float *b,int N){
    // 计算当前线程的全局索引
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // 计算网格总大小（总线程数）
    int stride = blockDim.x * gridDim.x;

    
}