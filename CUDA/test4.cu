
#include <hip/hip_runtime.h>
#include <stdio.h>
//CUDA�����,��װcuda��������
#define gpuErrchk(ans) {gpuAssert((ans),__FILE__, __LINE__);}

//������������
inline void gpuAssert(hipError_t code,const char *file,int line){
    if(code!=hipSuccess){
        fprintf(stderr,"GPU����:%s\n �ļ�:%s\n �к�:%d\n", hipGetErrorString(code),file,line);
        exit(code);
    }
}

// CPU��������ָ��ֵ��ʼ������
void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

// GPU�˺���������ִ�������ӷ�
// ʹ��������ѭ�����������
__global__ void addVectorsInto(float *result,float *a,float *b,int N){
    // ���㵱ǰ�̵߳�ȫ������
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // ���������ܴ�С�����߳�����
    int stride = blockDim.x * gridDim.x;

    // ʹ��������ѭ����ÿ���̴߳�����Ԫ��
    for(int i = index; i < N; i += stride){
        result[i] = a[i] + b[i];
    }
}

// CPU��������֤����Ƿ���ȷ
void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main(){
  // ���������С��2^21 = 2,097,152��Ԫ��
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  // ������������������ָ��
  float *a;
  float *b;
  float *c;

  // ����ͳһ�ڴ棬ʹCPU��GPU���ܷ���
  gpuErrchk(hipMallocManaged(&a, size));
  gpuErrchk(hipMallocManaged(&b, size));
  gpuErrchk(hipMallocManaged(&c, size));

  // ��ʼ�����飺aȫΪ3��bȫΪ4��cȫΪ0
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  // ����CUDA�˺���ִ������
  int threadsPerBlock = 256;
  // ������Ҫ�Ŀ�����ȷ���ܴ�������Ԫ��
  int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

  // �����˺������������ӷ�
  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
  // ���˺�����������
  gpuErrchk(hipPeekAtLastError());
  // �ȴ�GPU��ɲ��������ʱ����
  gpuErrchk(hipDeviceSynchronize());

  // ��֤���������Ԫ��Ӧ�õ���7��3+4��
  checkElementsAre(7, c, N);

  // �ͷ�ͳһ�ڴ�
  gpuErrchk(hipFree(a));
  gpuErrchk(hipFree(b));
  gpuErrchk(hipFree(c));

  return 0;
}